#include "hip/hip_runtime.h"
/*!
*******************************************************************************
* \file driver.cc
*
* \brief Implements driver for integrating guiding center trajectories
*
* \date Feb 2014
* \author C. Leland Ellison
*******************************************************************************
*/

#include <stdlib.h>
#include <iostream>
#include <Eigen/Dense>
#include <ctime> // For timing runtime
#include "input_parser.h"
#include "guiding_center.h"
#include "em_fields.h"
#include "axisymmetric_tokamak.h"
#include "integrator.h"
#include "runge-kutta4.h"
#include "noncanonical_symplectic.h"
#include "eigen_types.h"

#define GC_DIM 4 // Dimension of guiding center system

/*!
 * \brief Prints a line to standard out giving [time x[0] x[1] ....]
 *
 * @param[in] t Time
 * @param[in] x Position vector
 * @param[in] n_digits Number of digits to display in output
 */
void PrintState(double t, const Vector4 &x, int n_digits){
  // Formatting options: n_digits precision, all vector elements on one line
  static Eigen::IOFormat OneLineNDeep(n_digits, 0, "     ", "     ",
				      "", "", "", "");
  std::cout.setf( std::ios::fixed, std:: ios::floatfield );
  std::cout << t << "     " << x.format(OneLineNDeep) << std::endl;
}

/*!
 * \brief Body of the driver. Use program options to specify ode, integrator, dt, and n_steps.
 *
 */
int main(int argc, char *argv[]) {
  
  //// Read and extract input
  InputParser input_parser;
  int read_result;
  read_result = input_parser.ReadInput(argc, argv);
  if (read_result){return read_result; } // Quits on, e.g. --help

  // Initialize runtime parameters determined by input
  double dt, b0, r0, mu, newton_tolerance;
  int n_steps, save_nth, print_precision, max_iterations;
  std::vector<double> initial_conditions; 
  bool time_flag;
  std::string integrator_name;

  // Retrieve the values from the input parser
  input_parser.GetValue("dt", dt);
  input_parser.GetValue("n_steps", n_steps);
  input_parser.GetValue("save_nth", save_nth);
  input_parser.GetValue("initial_conditions", initial_conditions);
  input_parser.GetValue("time", time_flag);
  input_parser.GetValue("precision", print_precision);
  input_parser.GetValue("b0", b0);
  input_parser.GetValue("r0", r0);
  input_parser.GetValue("integrator", integrator_name);
  input_parser.GetValue("mu", mu);
  input_parser.GetValue("tol", newton_tolerance);
  input_parser.GetValue("max_iter", max_iterations);

  //// Initialize model and integrator
  AxisymmetricTokamak em_fields(b0, r0);
  GuidingCenter guiding_center((EMFields *) &em_fields, mu);
  Integrator *integrator;
  if (integrator_name.compare("rk4")==0){
    integrator = new RungeKutta4(dt, guiding_center);
  }
  else if (integrator_name.compare("ncsi")==0){
    integrator = new NoncanonicalSymplectic(dt, guiding_center, 
					    newton_tolerance, max_iterations);
  }
  else{
    std::cout << "Unrecognized integrator. Try rk4 or ncsi" 
	      << std::endl;
    return 1;
  }

  //// Initial conditions
  // Make a vector matching the size of the ode system
  Vector4 x; 
  int n_initial_conditions;
  
  // If no initial conditions specified or they are of the wrong dimension
  if((!initial_conditions.size()) || 
     (initial_conditions.size() % (GC_DIM) )){
    // Use the default initial conditions: Vector of ones
    n_initial_conditions = 1;
    for (int i = 0; i < x.size(); ++i) {
      initial_conditions.push_back(1.0);
    }
  }
  else{
   // Otherwise, we have more than one initial condition to simulate
   n_initial_conditions = initial_conditions.size()/GC_DIM;
  }

  //// Record time?
  std::clock_t run_time;
  if(time_flag){
    run_time = std::clock();
  }

  //// Time advance each initial condition
  for (int j = 0; j < n_initial_conditions; ++j){
    integrator->Reset(); // Resets temporary variables in integrators
    double t = 0;
    // Set initial condition
    for (int i=0; i<GC_DIM; ++i){
      x[i] = initial_conditions[j*GC_DIM + i];
    }
    PrintState(t, x, print_precision); // Print initial position
    // Run standard stepping
    for (int i = 1; i <= n_steps; ++i) {
      integrator->Step(t, x);
      if(!(i%save_nth)){
	PrintState( t, x, print_precision);
      }
    }
  }

  //// Print run time?
  if(time_flag){
    run_time = std::clock() - run_time;
    std::cout << "Run time: " << (double)run_time/CLOCKS_PER_SEC << std::endl;
  }

  //// Clean up
  delete integrator;
  return 0;
}
