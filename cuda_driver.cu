#include "hip/hip_runtime.h"
/*!
*******************************************************************************
* \file driver.cc
*
* \brief Implements driver for integrating guiding center trajectories
*
* \date Feb 2014
* \author C. Leland Ellison
*******************************************************************************
*/

#include <stdlib.h>
#include <iostream>
#include <Eigen/Dense>
#include <ctime> // For timing run time
#include "input_parser.h"
#include "guiding_center.h"
#include "em_fields.h"
#include "axisymmetric_tokamak.h"
#include "integrator.h"
#include "runge-kutta4.h"
#include "noncanonical_symplectic.h"
#include "eigen_types.h" // Typedef for Vector4, etc.
#include "cuda_error.h" // HANDLE_ERROR macro


#define GC_DIM 4 // Dimension of guiding center system


/*!
 * \brief Prints a line to standard out giving [time x[0] x[1] ....]
 *
 * @param[in] t Time
 * @param[in] x Position vector
 * @param[in] n_digits Number of digits to display in output
 */
void PrintState(double t, const Vector4 &x, int n_digits){
  // Formatting options: n_digits precision, all vector elements on one line
  static Eigen::IOFormat OneLineNDeep(n_digits, 0, "     ", "     ",
				      "", "", "", "");
  std::cout.setf( std::ios::fixed, std:: ios::floatfield );
  std::cout << t << "     " << x.format(OneLineNDeep) << std::endl;
}


// Kernel for time advance
// template <class I>
__global__ void step_positions(Vector4 *x, double t, const double kdt,  
			       const int kNSteps, const int kNParticles, 
			       const double kB0, const double kR0,
			       const double kMu){
  // Thread identification
  int idx=blockIdx.x*blockDim.x + threadIdx.x;

  // Integrator initialization
  AxisymmetricTokamak em_fields(kB0, kR0);
  GuidingCenter model((EMFields *) &em_fields, kMu);
  RungeKutta4 integrator(kdt, model);

  // Time advance
  for(int i=0; i<kNSteps; ++i){
    if (idx < kNParticles){
      integrator.Step(t, x[idx]);
    }
    __syncthreads(); // Likely not necessary, but doesn't slow down
  }
}


/*!
 * \brief Body of the driver. Use program options to specify ode, integrator, dt, and n_steps.
 *
 */
int main(int argc, char *argv[]) {
  
  //// Read and extract input
  InputParser input_parser;
  int read_result;
  read_result = input_parser.ReadInput(argc, argv);
  if (read_result){return read_result; } // Quits on, e.g. --help

  //// Retrieve the values from the input parser
  const double kdt = input_parser.GetValue<double>("dt");
  const int kNSteps = input_parser.GetValue<int>("n_steps");
  const int kSaveNth = input_parser.GetValue<int>("save_nth");
  std::vector<double> initial_conditions = 
    input_parser.GetValue<std::vector<double> >("initial_conditions");
  const bool kTimeFlag = input_parser.GetValue<bool>("time");
  const int kPrintPrecision = input_parser.GetValue<int>("precision");
  const double kB0 = input_parser.GetValue<double>("b0");
  const double kR0 = input_parser.GetValue<double>("r0");
  const std::string kIntegratorName = 
    input_parser.GetValue<std::string>("integrator");
  const double kMu = input_parser.GetValue<double>("mu");
  const double kSolveTolerance = input_parser.GetValue<double>("tol");
  const int kMaxIterations = input_parser.GetValue<int>("max_iter");
  const int kNParticles = input_parser.GetValue<int>("n_particles"); 
  const int kBlockSize = input_parser.GetValue<int>("block_size");  

  //// Check for valid parameters
  // Integrator
  if (kIntegratorName.compare("rk4") && kIntegratorName.compare("ncsi") ){
    std::cout << "Unrecognized integrator. Try rk4 or ncsi" << std::endl;
    return 1;
  }
  // Initial conditions
  if (initial_conditions.size() % (GC_DIM) ){
    std:: cout << "Wrong number of initial conditions. Use 4." << std::endl;
    return 1;
  }

  //// Set up CUDA parameters
  //       Should  check for cuda-capable device, here
  hipSetDevice(1); // Some people hop on first device while running use
  dim3 dimBlock(kBlockSize);
  dim3 dimGrid(ceil(kNParticles/(float)kBlockSize));
  
  //// Data set up
  // Host side
  Vector4 x_host[kNParticles];
  for(int i=0; i<kNParticles; ++i){
    for(int j=0; j<GC_DIM; ++j){
      x_host[i][j] = initial_conditions[j];
    }
  }
  double t = 0.0;
  // Device side
  Vector4 *x_device;
  HANDLE_ERROR( hipMalloc((void **)&x_device, 
			   kNParticles*sizeof(Vector4)) );
  HANDLE_ERROR( hipMemcpy(x_device, x_host, kNParticles*sizeof(Vector4), 
			  hipMemcpyHostToDevice) );

  //// Record time?
  std::clock_t run_time;
  if(kTimeFlag){
    run_time = std::clock();
  }

  // Print initial state
  PrintState(t, x_host[0], kPrintPrecision);

  //// Time advance
  for(int i=0; i<kNSteps/kSaveNth; ++i){
    // step_positions<RungeKutta4><<<dimGrid, dimBlock>>>(x_device, t, kdt,
    // 						      kSaveNth, kNParticles,
    // 						      kB0, kR0, kMu);
    step_positions<<<dimGrid, dimBlock>>>(x_device, t, kdt, kSaveNth, 
					  kNParticles, kB0, kR0, kMu);

    HANDLE_ERROR( hipGetLastError() ); // Check for kernel errors
    t += kSaveNth*kdt; // Advance host time
    // Advance host positions
    HANDLE_ERROR( hipMemcpy(x_host, x_device, kNParticles*sizeof(Vector4),
			     hipMemcpyDeviceToHost) );
    // Print state
    PrintState(t, x_host[0], kPrintPrecision);
  }

  //// Print run time?
  if(kTimeFlag){
    run_time = std::clock() - run_time;
    std::cout << "Run time: " << (double)run_time/CLOCKS_PER_SEC 
	      << std::endl;
  }

  //// Clean up
  HANDLE_ERROR( hipFree(x_device) );
  return 0;
}
