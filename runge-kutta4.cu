#include "hip/hip_runtime.h"
/*!
*******************************************************************************
* \file runge-kutta.cc
*
* \brief Implementation of Runge-Kutta4 - adapted from general version and for CUDA. 
*
* \date July 2014
* \author C. Leland Ellison
* \version 0.1
*
*******************************************************************************
*/

#include "runge-kutta4.h"

 /*!
  * 
  * \brief Constructor which implements common methods for convenience. Specify an order with an integer, and this constructor will set the corresponding coefficients.
  *
  * @param[in] kdt Numerical step size
  * @param[in] kGuidingCenter ODE system being solved
  * @param[in] kOrder Order of Runge-Kutta method. 2 and 4 are implemented.
  * 
  */
RungeKutta4::RungeKutta4(const double kdt, 
			 const GuidingCenter &kGuidingCenter) : 
  Integrator(kdt, kGuidingCenter), kOrder_(4) {
 
  //// Setting of the various coefficients
  a_(1,0) = 0.5;
  a_(2,1) = 0.5;
  a_(3,2) = 1.0;
  b_(0) = 1./6.;
  b_(1) = 1./3.;
  b_(2) = 1./3.;
  b_(3) = 1./6.;
  c_(1) = 0.5;
  c_(2) = 0.5;
  c_(3) = 1.0;
}

RungeKutta4::~RungeKutta4(){
}

/*!
 * \brief Explicit advance x(t) forward in time by step size kdt. RK:
 *        x_{n+1} = x_n + sum_{i=1}^s b_i k_i
 *            k_i = h f(t_n + c_i h, y_n + sum_{j=1}^{i-1} a_ij k_j 
 * 
 * @param[in, out] t Simulation time. Advanced by kdt_
 * @param[in, out] x Position. At in: x(t=t_k) At out: x(t=t_{k+1})
 */
int RungeKutta4::Step(double &t, Vector4 &x) {

  // Assign k_i values
  for (int i=0; i<k_.cols(); ++i){
    // Get temporary position
    xtemp_ = x;
    for (int j=0; j<i; ++j){
      xtemp_ += a_(i,j)*k_.col(j);
    }
    // Evaluate the rhs at x_temp to determine k
    kGuidingCenter_.VectorField(t + c_(i)*kdt_, xtemp_, ftemp_);
    k_.col(i) = kdt_*ftemp_;
  }

  // Update x
  x = x + k_*b_;
  // Update t
  t = t + kdt_;

  /////////////////// DOUBLE * COMPATIBLE VERSION ////////////////////////
  // //// RECALL: the i-th row, j-th column of matrix m is m[j*n_rows + i]
  // // Assign k_i values
  // for (int i=0; i<kOrder_; ++i){
  //   // Get temporary position
  //   for(int j=0; j<kDimen_; ++j){
  //     xtemp_[j] = x[j];
  //   }
  //   for (int j=0; j<i; ++j){
  //     for (int k=0; k<kDimen_; ++k){
  // 	xtemp_[k] += a_(i,j)*k_[j*kDimen_ + k]; // Right??
  //     }
  //   }
  //   // Evaluate the rhs at x_temp to determine k
  //   model_.VectorField(t + c_(i)*kdt_, xtemp_, ftemp_);
  //   for (int j=0; j<kDimen_; ++j){
  //     k_[i*kDimen_ + j] = kdt_*ftemp_[j];
  //   }
  // }

  // // Update x
  // for(int j=0; j<kOrder_; ++j){
  //   for(int i=0; i<kDimen_; ++i){
  //     // x^i = k^i_j*b^j
  //     x[i] += k_[j*kDimen_ + i]*b_[j]; // Unintuitive and likely slow
  //   }
  // }
  // // Update t
  // t = t + kdt_;

  // Finally, check theta coordinate for exceeding 2pi rad
  // Assumes coordinates are cylindrical! 
  double pi=3.141592653589793;
  if(std::abs(x[1]) > 2*pi){
    int sign;
    if(x[1]>0){
      sign=1;
    }
    else{
      sign=-1;
    }
    x[1] -= sign*2*pi;
  }

  return 0;
}
